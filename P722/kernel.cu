﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(double *results, const int *n, unsigned int size);

__global__ void sigma(double* results, const int* n) {
    int k = blockDim.x * blockIdx.x + threadIdx.x ;
    int nk = n[k];
    double sum = 0;
    for (int div = 1; div <= nk; div++) {
        if (nk % div == 0) {
            sum += div;
        }
    }
    results[k] = sum;
}

int main()
{
    const int arraySize = 512;
    int n[arraySize];
    double results[arraySize];
    for (int i = 0; i < arraySize; i++) {
        n[i] = i + 1;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(results, n, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    for (int k = 0; k < arraySize; k++) {
        printf("%d,%f\n",
            n[k], results[k]);
    }

    
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *results, const int *n, unsigned int size)
{
    int *dev_n = 0;
    double *dev_results = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_n, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_results, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_n, n, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    sigma<<<1, size>>>(dev_results, dev_n);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(results, dev_results, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_n);
    hipFree(dev_results);
    
    return cudaStatus;
}
